#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <chrono>
#include <random>

#include "Tensor.cuh"
#include "MNISTLoader.cuh"
#include "Layer.cuh"
#include "FullyConnected.cuh"
#include "ReLU.cuh"
#include "SoftmaxCrossEntropy.cuh"
#include "XORGenerator.cuh"
#include "Network.cuh"

using std::vector, std::cout, std::endl, std::string, std::unique_ptr;

void measureTensorSpeed();
void testMatrixMultiplication();
void testMatrixMultiplication2();
void testFullyConnectedForward();
void testFullyConnectedForward2();
void testTranspose();
void testMatrixRowSum();
void testMatrixColumnSum();
void testFullyConnectedBackward();
void testCuda();
void testMatrixMultiplicationGPU();
void compareCpuGpuMatrixMultiplication();
void testXORGenerator();
void testLayers();
void testSum();
void testNetwork();

int main() {
	// TODO: Add nodiscard / noexcept to certain functions
	testNetwork();
	//MNISTLoader trainData("MNIST Data\\train-images.idx3-ubyte", "MNIST Data\\train-labels.idx1-ubyte", 100); // TODO: Change to 60000
	//trainData.printData(0, 10);
	//testMatrixMultiplication();

	//FullyConnected<double> fc(5, 10, 0, Layer::WeightInitializationHeuristic::heNormal);
	//MNISTLoader testData("MNIST Data\\t10k-images.idx3-ubyte", "MNIST Data\\t10k-labels.idx1-ubyte", 10000);
}

// This is just a function for timing execution speeds. Used only for debugging.
void measureTensorSpeed() {
    using std::chrono::high_resolution_clock;
    using std::chrono::duration;

    Tensor<double> toTest({ 10,10,10 });

    auto t1 = high_resolution_clock::now();
    for (int i = 0; i < 1000000; i++) {
        toTest.set({ 2,2,2 }, 1.1);
    }
    auto t2 = high_resolution_clock::now();

    /* Getting number of milliseconds as a double. */
    duration<double, std::milli> ms_double = t2 - t1;
    std::cout << ms_double.count() << "ms\n";
}

void testMatrixMultiplication() {
	Tensor<double> t1({ 2,3 });
	t1.set({ 0,0 }, 0);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<double> t2({ 3,1 });
	t2.set({ 0,0 }, 1.5);
	t2.set({ 1,0 }, 3);
	t2.set({ 2,0 }, 5.5);

	Tensor<double> t3 = t1.matrixMultiply(t2);
	// [14.55; 40.125] expected
	assert(t3.get({ 0 }) == 14.55 && t3.get({ 1 }) == 40.125);
}

void testMatrixMultiplication2() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	Tensor<double> t1({ 32,5000 });
	t1.set({ 0,0 }, 0);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<double> t2({ 5000,2500 });
	t2.set({ 0,0 }, 1.5);
	t2.set({ 1,0 }, 3);
	t2.set({ 2,0 }, 5.5);
	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 10; i++) {
		Tensor<double> t3 = t1.matrixMultiply(t2);
		//Tensor<double> t3 = t1.matrixMultiply<double>(t2);
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

// TODO: Actually make a test case that asserts if the output is correct
void testFullyConnectedForward() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;

	FullyConnected<double> fc(2, 1, 0.01);

	Tensor<double> t2({ 1,2 });
	t2.set({ 0,0 }, 2);
	t2.set({ 0,1 }, 3);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 100000; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		auto t3 = fc.forward(t2);
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void testFullyConnectedForward2() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;

	FullyConnected<double> fc(5000, 5000, 0.001);

	Tensor<double> t2({ 32,5000 });
	Tensor<double> t3({ 32,5000 });
	std::uniform_real_distribution<double> uniform(0.0, static_cast<double>(sqrt(2.0 / 10)));
	std::default_random_engine engine(1);
	t2.setToRandom(uniform, engine);
	t3.setToRandom(uniform, engine);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 10; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		fc.forward(t2);
		fc.backward(t3);
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}


void testTranspose() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	Tensor<double> t1({ 5000,2500 });
	t1.set({ 0,0 }, 3.2);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<float> t2({ 3 });
	t2.set({ 0 }, 1.5);
	t2.set({ 1 }, 3.5);
	t2.set({ 2 }, 4.0);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 100; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		auto t3 = t1.transpose();
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void testMatrixRowSum() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	Tensor<double> t1({ 32,5000 });
	t1.set({ 0,0 }, 3.2);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<float> t2({ 3 });
	t2.set({ 0 }, 1.5);
	t2.set({ 1 }, 3.5);
	t2.set({ 2 }, 4.0);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 1000; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		auto t3 = t1.matrixRowSum();
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void testMatrixColumnSum() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	Tensor<double> t1({ 32,5000 });
	t1.set({ 0,0 }, 3.2);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<float> t2({ 3 });
	t2.set({ 0 }, 1.5);
	t2.set({ 1 }, 3.5);
	t2.set({ 2 }, 4.0);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 1000; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		auto t3 = t1.matrixColumnSum();
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void testFullyConnectedBackward() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;

	FullyConnected<double> fc(2, 1, 0,01);

	Tensor<double> t2({ 1,2 });
	t2.set({ 0,0 }, 2);
	t2.set({ 0,1 }, 3);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 100000; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		auto t3 = fc.forward(t2);
		auto t4 = fc.backward(t2);
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void testCuda() {
	// Initialize arrays A, B, and C.
	//double A[3], B[3], C[3];

	// Populate arrays A and B.
	//A[0] = 5; A[1] = 8; A[2] = 3;
	//B[0] = 7; B[1] = 6; B[2] = 4;

	// Sum array elements across ( C[0] = A[0] + B[0] ) into array C using CUDA.
	//TensorMathGPU::kernel(A, B, C, 3);

	// Print out result.
//	std::cout << "C = " << C[0] << ", " << C[1] << ", " << C[2] << std::endl;
}

void testMatrixMultiplicationGPU() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	Tensor<double> t1({ 2,3 });
	t1.set({ 0,0 }, 0);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<double> t2({ 3,1 });
	t2.set({ 0,0 }, 1.5);
	t2.set({ 1,0 }, 3);
	t2.set({ 2,0 }, 5.5);
	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 10; i++) {
		Tensor<double> t3 = t1.matrixMultiplyGPU(t2);
		//Tensor<double> t3 = t1.matrixMultiply<double>(t2);
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void compareCpuGpuMatrixMultiplication() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	using std::uniform_real_distribution;
	using std::default_random_engine;

	uniform_real_distribution<double> dist(-1, 1);
	default_random_engine engine(0);

	Tensor<double> t1({ 32,5000 });
	t1.setToRandom(dist, engine);
	Tensor<double> t2({ 5000,5000 });
	t2.setToRandom(dist, engine);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 10; i++) {
		Tensor<double> t3 = t1.matrixMultiply(t2);
	}

	auto time2 = high_resolution_clock::now();
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << "CPU: " << ms_double.count() << "ms\n";

	auto time3 = high_resolution_clock::now();
	for (int i = 0; i < 10; i++) {
		Tensor<double> t3 = t1.matrixMultiplyGPU(t2);
	}

	auto time4 = high_resolution_clock::now();
	duration<double, std::milli> ms_double2 = time4 - time3;
	std::cout << "GPU: " << ms_double2.count() << "ms\n";
}

//void testXORGenerator() {
//	using std::default_random_engine;
//	default_random_engine randomEngine(0);
//
//	XORGenerator<double> generator(0);
//	generator.generate(64);
//	vector<int> indices = generator.getInputs().getRandomIndices(32, randomEngine);
//	Tensor<double> inputBatch = generator.getInputs().getBatch(indices);
//	Tensor<double> outputBatch = generator.getLabels().getBatch(indices);
//}

void testLayers() {
	FullyConnected<double> fc(100,100,0.01);
	ReLU<double> relu;
	SoftmaxCrossEntropy<double> softmax;
}

void testSum() {
	using std::chrono::high_resolution_clock;
	using std::chrono::duration;
	Tensor<double> t1({ 32,5000 });
	t1.set({ 0,0 }, 3.2);
	t1.set({ 0,1 }, 1);
	t1.set({ 0,2 }, 2.1);
	t1.set({ 1,0 }, 1.5);
	t1.set({ 1,1 }, 3);
	t1.set({ 1,2 }, 5.25);

	Tensor<float> t2({ 3 });
	t2.set({ 0 }, 1.5);
	t2.set({ 1 }, 3.5);
	t2.set({ 2 }, 4.0);

	auto time1 = high_resolution_clock::now();
	for (int i = 0; i < 1000; i++) {
		//Tensor<double> t3 = t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAdd<double>(t2);
		//t1.elementwiseAddInPlace(t2);
		auto t4 = t1.sum(0);
	}

	auto time2 = high_resolution_clock::now();

	/* Getting number of milliseconds as a double. */
	duration<double, std::milli> ms_double = time2 - time1;
	std::cout << ms_double.count() << "ms\n";
}

void testNetwork() {
	XORGenerator<double> generator(0);
	generator.generate(1024);

	default_random_engine randomEngine(0);

	vector<unique_ptr<Layer<double>>> layers;
	layers.push_back(std::make_unique<FullyConnected<double>>(2, 11, 0.01));
	layers.push_back(std::make_unique<ReLU<double>>());
	layers.push_back(std::make_unique<FullyConnected<double>>(11, 2, 0.01));
	layers.push_back(std::make_unique<SoftmaxCrossEntropy<double>>());

	Network network(std::move(layers));
	for (int i = 0; i < 10000; i++) {
		vector<int> indices = generator.getInputs().getRandomIndices(64, randomEngine);
		Tensor<double> inputs = generator.getInputs().getBatch(indices);
		Tensor<int> labels = generator.getLabels().getBatch(indices);

		Tensor<double> predicted = network.forward(inputs);
		if (i % 500 == 0) {
			cout << "Loss " << i << ": " << network.loss(predicted, labels) << endl;
		}
		network.backward(labels);
	}
}